#include <stdio.h>
#include <hip/hip_runtime.h>

#define TILE 16

__global__ void matmul_kernel_tiled(double *A, double *B, double *C, int N){
    __shared__ double sA[TILE][TILE];
    __shared__ double sB[TILE][TILE];

    int row = blockIdx.y*TILE + threadIdx.y;
    int col = blockIdx.x*TILE + threadIdx.x;

    double sum=0.0;

    for(int t=0; t<(N+TILE-1)/TILE; t++){
        if(row<N && t*TILE+threadIdx.x<N)
            sA[threadIdx.y][threadIdx.x] = A[row*N + t*TILE + threadIdx.x];
        else
            sA[threadIdx.y][threadIdx.x] = 0.0;

        if(col<N && t*TILE+threadIdx.y<N)
            sB[threadIdx.y][threadIdx.x] = B[(t*TILE+threadIdx.y)*N + col];
        else
            sB[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads();

        for(int k=0;k<TILE;k++)
            sum += sA[threadIdx.y][k]*sB[k][threadIdx.x];

        __syncthreads();
    }

    if(row<N && col<N)
        C[row*N + col] = sum;
}

int main(int argc,char** argv){
    int N=1024;
    if(argc>1) N=atoi(argv[1]);

    size_t bytes = N*N*sizeof(double);
    double *h_A = (double*)malloc(bytes);
    double *h_B = (double*)malloc(bytes);
    double *h_C = (double*)malloc(bytes);

    // Matching OpenMP/MPI initialization
    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            h_A[i*N + j] = i + j*0.5;
            h_B[i*N + j] = i - j*0.5;
        }
    }

    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes); hipMalloc(&d_B, bytes); hipMalloc(&d_C, bytes);
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    dim3 block(TILE,TILE);
    dim3 grid((N+TILE-1)/TILE,(N+TILE-1)/TILE);

    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start);

    matmul_kernel_tiled<<<grid,block>>>(d_A,d_B,d_C,N);

    hipError_t err = hipGetLastError();
    if(err != hipSuccess){
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
        return -1;
    }

    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms; hipEventElapsedTime(&ms,start,stop);
    hipMemcpy(h_C,d_C,bytes,hipMemcpyDeviceToHost);

    printf("CUDA MatMul (Tiled): N=%d grid=(%d,%d) block=(%d,%d) Time=%f ms Sample C[0][0]=%f\n",
           N, grid.x, grid.y, block.x, block.y, ms, h_C[0]);

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);

    return 0;
}
